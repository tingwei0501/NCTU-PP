/**********************************************************************
 * DESCRIPTION:
 *   Wave Equation - cu Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <time.h>
 
 #define MAXPOINTS 1000000
 #define MAXSTEPS 1000000
 #define MINPOINTS 20
 #define PI 3.14159265
 
 void check_param(void);
 void init_line(void);
 void update (void);
 void printfinal (void);
 
 int nsteps,                 	/* number of time steps */
     tpoints, 	     		/* total points along string */
     rcode;                  	/* generic return code */
 float  values[MAXPOINTS+2], 	/* values at time t */
        oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
        newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
 
 
 /**********************************************************************
  *	Checks input values from parameters
  *********************************************************************/
 void check_param(void)
 {
    char tchar[20];
 
    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
       printf("Enter number of points along vibrating string [%d-%d]: "
            ,MINPOINTS, MAXPOINTS);
       scanf("%s", tchar);
       tpoints = atoi(tchar);
       if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
          printf("Invalid. Please enter value between %d and %d\n", 
                  MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
       printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
       scanf("%s", tchar);
       nsteps = atoi(tchar);
       if ((nsteps < 1) || (nsteps > MAXSTEPS))
          printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
 
    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
 
 }

 /**********************************************************************
  *     Initialize points on line
  *********************************************************************/
 __global__ void init_line(int tpoints, float *GPU_values, float *GPU_oldval)
 {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Calculate initial values based on sine curve */
     
    /* Initialize old values array */
    GPU_values[id] = sin((2.0 * PI)*((float)id/(float)(tpoints-1)));
    GPU_oldval[id] = sin((2.0 * PI)*((float)id/(float)(tpoints-1)));
 }
 
 /**********************************************************************
  *      Calculate new values using wave equation
  *********************************************************************/
 
 /**********************************************************************
  *     Update all values along line a specified number of times
  *********************************************************************/

 __global__ void update(int tpoints, int nsteps, float *GPU_values, float *GPU_oldval, float *GPU_newval)
 {
    int i;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Update values for each time step */
    for (i = 1; i<= nsteps; i++) {
       /* Update points along line for this time step */
       if ((id == 0) || (id  == tpoints-1))
            GPU_newval[id] = 0.0;
       else 
            GPU_newval[id] = 1.82 * GPU_values[id] - GPU_oldval[id];
 
       /* Update old values with new values */
       GPU_oldval[id] = GPU_values[id];
       GPU_values[id] = GPU_newval[id];
    }
 }
 
 /**********************************************************************
  *     Print final results
  *********************************************************************/
 void printfinal()
 {
    int i;
 
    for (i = 0; i < tpoints; i++) {
       printf("%6.4f ", values[i]);
       if (i%10 == 9)
          printf("\n");
    }
 }
 
 /**********************************************************************
  *	Main program
  *********************************************************************/
 int main(int argc, char *argv[])
 {
     sscanf(argv[1],"%d",&tpoints);
     sscanf(argv[2],"%d",&nsteps);
     check_param();

     float *GPU_values, *GPU_oldval, *GPU_newval;
     // GPU memory
     hipMalloc(&GPU_values, sizeof(values));
     hipMalloc(&GPU_oldval, sizeof(values));
     hipMalloc(&GPU_newval, sizeof(values));

     printf("Initializing points on the line...\n");
     init_line<<<((tpoints + 1023) >> 10), 1024>>>(tpoints, GPU_values, GPU_oldval);

     printf("Updating all points for all time steps...\n");
     // <<<numBlocks, threadsPerBlock>>>
     update<<<((tpoints + 1023) >> 10), 1024>>>(tpoints, nsteps, GPU_values, GPU_oldval, GPU_newval);
     printf("Printing final results...\n");
     hipMemcpy(values, GPU_values, sizeof(values), hipMemcpyDeviceToHost);
     printfinal();
     printf("\nDone.\n\n");
     
     return 0;
 }
 